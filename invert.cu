#include "hip/hip_runtime.h"
The goal of this assignment is for you to develop a program that highlights something that is very interesting or applicable to your personal goals for this course. You can use any lab from the course that is CUDA/GPU-based or any other development environment that you want to use, as long as you make it clear in README.md, other documentation, and capstone project demonstration/presentation. You have complete freedom to use the knowledge that have gained from any GPU-based material (so no simple C/C++ or pythonn multithreaded code) that we have covered or that you have learned independent of the material from the lessons in this course. 

If you develop outside of the Coursera Labs that exist for this course, you can program in other languages than C/C++, such as Python, Java, JavaScript, etc.  Also you can use frameworks/libraries that utilize GPUs under the hood, such as PyCuda, JCuda, TensorFlow, Torch, etc. You will need to make it clear in your material how you install any languages, libraries, etc. and how you specifically use GPU hardware. You can also use non-Nvidia GPUs, though that will mean that you may need to learn non-CUDA-based language/library constructs/syntax.

Have fun with this project and make sure you get the most out of this. Feel free to spend as much time as you wish on making this project the highlight of your work in this specialization, but you should spend a minimum of 8 hours on this work. The development of this final course and all assignments was built around giving you extra time to work on this project.

You will be graded by your peers based on the following submissions:

Url pointing to a code repository (GitHub, GitLab, BitBucket, Google Drive, Box, etc. as long as it is publicly available) holding the code for the assignment. A template project can be found at 
https://github.com/PascaleCourseraCourses/CUDAatScaleForTheEnterpriseCourseProjectTemplate
.

Proof of code execution, in the form of images, csv data files (input and output), log files, etc.

Short text description of your work.

Project Presentation/Demonstration (5-10 minutes)

Note: You will need to bring data into and out of the laboratory environment, along with any code developed along the way, so any output data will need to be stored in the code repository and committed/pushed back out to GitHub, Gitlab, etc. 

Listed below are data sources for images are listed below:

USC Viterbi School of Engineering's SIPI Image Database - 
https://sipi.usc.edu/database/database.php

UC Irvine Machine Learning Repository (Iris, CMU Face, and MNist Handwritten Digits are image databases) - 
https://archive-beta.ics.uci.edu

Creative Commons Search Tool - Will need to download individually but a wide variety of images with flexible licenses that will allow for use in this course -
https://search.creativecommons.org

Aquila - a C++ library for processing audio, including .wav files, into signals data that can be used in conjunction with the signal processing capabilities of NPP - 
http://aquila-dsp.org

Stanford's Synthesis Toolkit - C++ library for signal processing and generating synthetic audio files -
https://ccrma.stanford.edu/software/stk/

Example signals for different instruments - 
https://www.dsprelated.com/freebooks/pasp/Sound_Examples.html

Rubric:


